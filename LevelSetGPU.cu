#include "hip/hip_runtime.h"



///////////////////////////////// INCLUDES /////////////////////////////////
#include "LevelSetGPU.cuh"




/////////////////  FUNCTION IMPLEMENTATIONS : Wrappers around GPU Kernel functions //////////


void LevelSetCUDA::_setup(LevelSet* hostLS)
{

	 // copy parameters from one structure to another
	 // Ideally, these parameters would be in constant memory on GPU rather than on host.
	 //  For now, still just a copy from host to host 
	 Nx = hostLS->Nx;
	 Ny = hostLS->Ny;
	 dx = hostLS->dx;
	 dy = hostLS->dy;
	Xmin = hostLS->Xmin;
	Ymin = hostLS->Ymin;


    // Determine block and thread amounts for 2D kernel function
    int grid_dimX = ((Nx + BLOCK_SIZE-1) / BLOCK_SIZE) ;
    int grid_dimY = ((Ny + BLOCK_SIZE-1) / BLOCK_SIZE) ;
    dim3 grid(grid_dimX , grid_dimY);
    dim3 block(BLOCK_SIZE , BLOCK_SIZE);
	dimGrid.x=grid.x;
	dimGrid.y=grid.y;
	dimGrid.z=grid.z;
	dimBlock.x=block.x;
	dimBlock.y=block.y;
	dimBlock.z=block.z;



	HANDLE_ERROR( hipMalloc((void**)&Phi,  Ny*Nx*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc((void**)&Psi,  Ny*Nx*sizeof(float) ) );
	HANDLE_ERROR( hipMemcpy(Phi, hostLS->PhiGPU,  Ny*Nx*sizeof(float), hipMemcpyHostToDevice) );

//    	size_t PhiPitchInbytes;
//    	size_t PsiPitchInbytes;

//	// Allocate memory on device 
//	HANDLE_ERROR( hipMallocPitch((void**)&Phi, &PhiPitchInbytes, Ny, Nx) );
//	HANDLE_ERROR( hipMallocPitch((void**)&Psi, &PsiPitchInbytes, Ny , Nx));
//	PhiPitch = (int)(PhiPitchInbytes / sizeof(float));
//	PsiPitch = (int)(PsiPitchInbytes / sizeof(float));
//	
//    	// Copy data from host to device
//	HANDLE_ERROR( hipMemcpy2D(Phi, PhiPitchInbytes, hostLS->PhiGPU, 
//				Ny*sizeof(float), Ny*sizeof(float), Nx,
//					hipMemcpyHostToDevice) );



    // Allocate memory on device for vectors of points on implicit curve
	HANDLE_ERROR( hipMalloc((void**)&xi, (Nx*4+1)*sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&yj, (Nx*4+1)*sizeof(float)) );
	// Copy data from host to device
	HANDLE_ERROR( hipMemcpy(xi, hostLS->xi, (Nx*4+1)*sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(yj, hostLS->yj, (Nx*4+1)*sizeof(float), hipMemcpyHostToDevice) );




	printf("\nSetup complete. dx= %3.3f\n", dx );

 
}//_setup



void LevelSetCUDA::_transferPhiToHost(LevelSet* hostLS)
{
	//size_t PhiPitchInbytes = (size_t)(PhiPitch * sizeof(float));

    // Copy data back to host
	//HANDLE_ERROR( hipMemcpy2D(hostLS->PhiGPU, Ny*sizeof(float), Phi, PhiPitchInbytes, 
	//		Ny*sizeof(float), Nx, hipMemcpyDeviceToHost) );
			
	HANDLE_ERROR( hipMemcpy(hostLS->PhiGPU, Phi, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost) );		

}//_transferPhiToHost



void LevelSetCUDA::_teardown(LevelSet* hostLS)
{
	//size_t PhiPitchInbytes = (size_t)(PhiPitch * sizeof(float));

    // Copy data back to host
    //HANDLE_ERROR( hipMemcpy2D(hostLS->PhiGPU, Ny*sizeof(float), Phi, PhiPitchInbytes, 
		//	Ny*sizeof(float), Nx, hipMemcpyDeviceToHost) );
			
   HANDLE_ERROR( hipMemcpy(hostLS->PhiGPU, Phi, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost)  );						

    // Deallocate memory on device 
    HANDLE_ERROR(hipFree(Phi));
    HANDLE_ERROR(hipFree(Psi));



	printf("\nTeardown complete. \n " );

}//_tearDown




void LevelSetCUDA::_launchSimpleSignKernel( )
{
	
	printf("\nLaunching sign function kernel ...\n");

	int PhiPitch = Nx;

	testkernel1<<<dimGrid, dimBlock>>>(Phi, Nx, Ny, PhiPitch);//marks values -1 if <0 and 1 if >0
	HANDLE_ERROR( hipDeviceSynchronize() );


}//_launchSimpleSignKernel



void LevelSetCUDA::_getSignedDistanceFunction( )
{
	
	printf("Initializing levelset data from implicit curve\n");

	int PhiPitch = Nx;

	SetPhi<<<dimGrid, dimBlock>>>(Phi, Nx, Ny, PhiPitch, Nx*4, xi, yj, Xmin, Ymin, dx, dy);
	HANDLE_ERROR( hipDeviceSynchronize() );


}//_getSignedDistanceFunction





void LevelSetCUDA::_reinitFastMarchNewton(LevelSet* hostLS )
{
	
	printf("Reinitizing levelset using Fast Marching with Newton solver\n");


	//dim3 dimBlock2(8,16);
	//dim3 dimGrid2((Nx-1)/8 + 1,(Ny-1)/16 + 1);

    //int* ListVoxels = new int[Nx*Ny];     // use for debugging
    //float* ListCoeff = new float[16*Nx*Ny]; //only need these on gpu
    int* devListVoxels;
    float* devListCoeff;
    int numberOfVox = Nx*Ny;//4096;
    HANDLE_ERROR( hipMalloc((void**)&devListVoxels, numberOfVox*sizeof(int)) );
    HANDLE_ERROR( hipMemset(devListVoxels,0,sizeof(int)));    
    HANDLE_ERROR( hipMalloc((void**)&devListCoeff, 16*numberOfVox*sizeof(float) ) );

    //copy shape from host_phi temporarily to dev_psi before reinitializing on GPU
    //size_t PsiPitchInbytes = (size_t)(PsiPitch * sizeof(float));
    //HANDLE_ERROR( hipMemcpy2D(Psi, PsiPitchInbytes, hostLS->PhiGPU, Ny*sizeof(float),
			//	 Ny*sizeof(float), Nx, hipMemcpyHostToDevice) ); 
			

   HANDLE_ERROR( hipMemcpy(Psi, hostLS->PhiGPU, Ny*Nx*sizeof(float), hipMemcpyHostToDevice)  );

    printf("Starting to get voxels and coefficients\n");
    
    int PsiPitch = Nx;
    int PhiPitch = Nx;

    // get interpolating polynomial coefficients and the active set voxels
    getVoxels<<<dimGrid, dimBlock>>>(Psi, PsiPitch, devListVoxels, devListCoeff, Nx, Ny);   
    hipDeviceSynchronize();

    printf("Finished getting voxels and coefficients\n");

    // re-initialize each grid point
    float thres = 0.0001;
    
    reinitPhi<<<dimGrid, dimBlock>>>(Phi, PhiPitch, Psi, devListVoxels, devListCoeff, Nx, Ny, dx, thres);
    hipDeviceSynchronize() ;

    HANDLE_ERROR(hipFree(devListVoxels));
    HANDLE_ERROR(hipFree(devListCoeff));

}//_reinitFastMarchNewton



void LevelSetCUDA::_extendVelocityF(LevelSet* hostLS )
{
	
	printf("Extending velocity field F(x,y) from levelset outward.\n");

    	//size_t FextPitchInbytes;
    	//size_t AccptPitchInbytes;


 	HANDLE_ERROR( hipMalloc((void**)&Fspeed, Ny*Nx*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc((void**)&Accept,  Ny*Nx*sizeof(float) ));
	HANDLE_ERROR( hipMemset(Accept,0,sizeof(int)));    
	HANDLE_ERROR( hipMalloc((void**)&Ftemp,  Ny*Nx*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc((void**)&Atemp,  Ny*Nx*sizeof(float) ));
	HANDLE_ERROR( hipMemset(Atemp,0,sizeof(int))); 
	int PhiPitch = Nx;
	int AccptPitch = Nx;
	

	// Allocate memory on device 
//	HANDLE_ERROR( hipMallocPitch((void**)&Fspeed, &FextPitchInbytes, Ny, Nx) );
//	HANDLE_ERROR( hipMallocPitch((void**)&Accept, &AccptPitchInbytes, Ny , Nx));
//	HANDLE_ERROR( hipMemset(Accept,0,sizeof(int)));    
//	HANDLE_ERROR( hipMallocPitch((void**)&Ftemp, &FextPitchInbytes, Ny, Nx) );
//	HANDLE_ERROR( hipMallocPitch((void**)&Atemp, &AccptPitchInbytes, Ny , Nx));
//	HANDLE_ERROR( hipMemset(Atemp,0,sizeof(int))); 
//	int AccptPitch = (int)(AccptPitchInbytes / sizeof(int));

	FastMarchInit<<<dimGrid, dimBlock>>>(Phi, PhiPitch, Accept, AccptPitch, Fspeed, Nx, Ny, dx, dy, Xmin, Ymin);
	HANDLE_ERROR( hipDeviceSynchronize() );
	hipDeviceSynchronize();
	
	
	//printf("\n%d,  %d\t", PhiPitch, AccptPitch);
	

    	int   * newA, * oldA;
     	float * newF, * oldF;  
	
	int count;
	bool toggle=true;
	int countMax = Nx*floor(log(Nx)); //Not sure this is the upperbound. Best guess based on algorithm.
	for (count=1 ; count<countMax ; ++count){
		if (toggle){
		toggle = false;
		oldA = Accept;
		newA = Atemp;
		oldF = Fspeed;
		newF = Ftemp;
		} else {
		toggle = true;
		oldA = Atemp;
		newA = Accept;
		oldF = Ftemp;
		newF = Fspeed;		
		}
	
		FastMarchVelocity<<<dimGrid, dimBlock>>>(count, Phi, PhiPitch, oldA, AccptPitch, oldF, newA, newF, Nx, Ny, dx, dy);
		HANDLE_ERROR( hipDeviceSynchronize() );
		hipDeviceSynchronize(); //needed this to synchronize correctly between iterations		
	} 
	hipDeviceSynchronize() ;
	
	


      HANDLE_ERROR( hipMemcpy(hostLS->PhiGPU, Phi, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost)  );
      HANDLE_ERROR( hipMemcpy(hostLS->Fext, newF, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost)  );
      HANDLE_ERROR( hipMemcpy(hostLS->Accept, newA, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost)  );

//     HANDLE_ERROR( hipMemcpy(hostLS->PhiGPU, Phi, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost)  );
//     HANDLE_ERROR( hipMemcpy(hostLS->Fext, Fspeed, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost)  );
//     HANDLE_ERROR( hipMemcpy(hostLS->Accept, Accept, Ny*Nx*sizeof(float), hipMemcpyDeviceToHost)  );
	
	
//    size_t PhiPitchInbytes = (size_t)(PhiPitch * sizeof(float));

//    // Copy data back to host
//    HANDLE_ERROR( hipMemcpy2D(hostLS->PhiGPU, Ny*sizeof(float), Phi, PhiPitchInbytes, 
//			Ny*sizeof(float), Nx, hipMemcpyDeviceToHost) );
//			
//    // Copy data back to host
//    HANDLE_ERROR( hipMemcpy2D(hostLS->Fext, Ny*sizeof(float), Fspeed, FextPitchInbytes, Ny*sizeof(float), Nx, hipMemcpyDeviceToHost) );
//    //HANDLE_ERROR( hipMemcpy2D(hostLS->h_Fext, Ny*sizeof(float), newF, FextPitchInbytes, Ny*sizeof(float), Nx, hipMemcpyDeviceToHost) );
//    
//			
//    // Copy data back to host
//    HANDLE_ERROR( hipMemcpy2D(hostLS->Accept, Ny*sizeof(int), Accept, AccptPitchInbytes, Ny*sizeof(int), Nx, hipMemcpyDeviceToHost) );		
//    //HANDLE_ERROR( hipMemcpy2D(hostLS->h_Accept, Ny*sizeof(int), newA, AccptPitchInbytes, Ny*sizeof(int), Nx, hipMemcpyDeviceToHost) );
//    	
	
    HANDLE_ERROR(hipFree(Fspeed));
    HANDLE_ERROR(hipFree(Accept));	
    HANDLE_ERROR(hipFree(Ftemp));
    HANDLE_ERROR(hipFree(Atemp));	
	


}//_extendVelocityF











